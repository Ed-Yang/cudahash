
// #include <cuda.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
    // printf("out = %f\n", out[0]);
}

void run_vector_add(float *out, float *a, float *b, int n)
{
    float *d_a, *d_b, *d_out; 

    hipMalloc((void**)&d_a, sizeof(float) * n);
    hipMalloc((void**)&d_b, sizeof(float) * n);
    hipMalloc((void**)&d_out, sizeof(float) * n);
    
    hipMemcpy(d_a, a, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * n, hipMemcpyHostToDevice);

    vector_add<<<1,1>>>(d_out, d_a, d_b, n);

    hipMemcpy(out, d_out, sizeof(float) * n, hipMemcpyDeviceToHost);

    printf(">>> n = %d\n", n);
    for (int i=0; i < n; i++)
        printf(">>> i=%d, out = %f\n", i, out[i]);

}
